#include "hip/hip_runtime.h"
#include "mdct_cuda.hpp"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <float.h>
#include <iostream>

#ifndef S_MUL
#define S_MUL(a, b) ((a) * (b))
#endif

#define CHECK_CUDA_ERROR(call)                                                 \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

#define CHECK_LAST_CUDA_ERROR()                                                \
  do {                                                                         \
    hipError_t err = hipGetLastError();                                      \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__,         \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// CUDA kernel
__global__ void doPreRotation(const var_t *xp1, var_t *yp, const var_t *t,
                              int N4, int shift, int stride, int N2,
                              var_t sine) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N4) {
    const var_t *xp1_i = xp1 + i * 2 * stride;
    const var_t *xp2_i = xp1 + stride * (N2 - 1) - i * 2 * stride;

    var_t yr, yi;
    yr = -S_MUL(*xp2_i, t[i << shift]) + S_MUL(*xp1_i, t[(N4 - i) << shift]);
    yi = -S_MUL(*xp2_i, t[(N4 - i) << shift]) - S_MUL(*xp1_i, t[i << shift]);

    yp[i * 2] = yr - S_MUL(yi, sine);
    yp[i * 2 + 1] = yi + S_MUL(yr, sine);
  }
}

void preRotateWithCuda(const var_t *host_xp1, var_t *host_yp,
                       const var_t *host_t, int N, int shift, int stride,
                       var_t sine) {
  int N2 = N >> 1;
  int N4 = N >> 2;
  var_t *dev_xp1;
  var_t *dev_yp;
  var_t *dev_t;

  hipMalloc((void **)&dev_xp1, N4 * 2 * stride * sizeof(var_t));
  hipMalloc((void **)&dev_yp, N4 * 2 * sizeof(var_t));
  hipMalloc((void **)&dev_t, (N4 << shift) * sizeof(var_t));

  hipMemcpy(dev_xp1, host_xp1, N4 * 2 * stride * sizeof(var_t),
             hipMemcpyHostToDevice);
  hipMemcpy(dev_t, host_t, (N4 << shift) * sizeof(var_t),
             hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N4 + blockSize - 1) / blockSize;

  doPreRotation<<<numBlocks, blockSize>>>(dev_xp1, dev_yp, dev_t, N4, shift,
                                          stride, N2, sine);

  hipDeviceSynchronize();
  hipMemcpy(host_yp, dev_yp, N4 * 2 * sizeof(var_t), hipMemcpyDeviceToHost);

  hipFree(dev_xp1);
  hipFree(dev_yp);
  hipFree(dev_t);
}

__global__ void postAndMirrorKernel(var_t *d_out, const var_t *t,
                                    const var_t *window, int N2, int N4,
                                    int shift, var_t sine, int overlap) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Handle post-rotation part
  if (idx < (N4 + 1) >> 1) {
    var_t re, im, yr, yi;
    var_t t0, t1;

    // Calculate left pointer position
    var_t *yp0 = d_out + (overlap >> 1) + 2 * idx;
    // Calculate right pointer position
    var_t *yp1 = d_out + (overlap >> 1) + N2 - 2 - 2 * idx;

    // Process the first pair of values
    re = yp0[0];
    im = yp0[1];
    t0 = t[idx << shift];
    t1 = t[(N4 - idx) << shift];
    yr = S_MUL(re, t0) - S_MUL(im, t1);
    yi = S_MUL(im, t0) + S_MUL(re, t1);

    // Save the first pair of results
    var_t yr1 = yr;
    var_t yi1 = yi;

    // Process the second pair of values
    re = yp1[0];
    im = yp1[1];
    t0 = t[(N4 - idx - 1) << shift];
    t1 = t[(idx + 1) << shift];
    yr = S_MUL(re, t0) - S_MUL(im, t1);
    yi = S_MUL(im, t0) + S_MUL(re, t1);

    // Write results in the same order as the CPU version
    yp0[0] = -(yr1 - S_MUL(yi1, sine)); // Left real
    yp1[1] = yi1 + S_MUL(yr1, sine);    // Right imag
    yp1[0] = -(yr - S_MUL(yi, sine));   // Right real
    yp0[1] = yi + S_MUL(yr, sine);      // Left imag
  }

  // sync
  __syncthreads();

  // Handle mirror part
  // Use a different index for the mirror operation to ensure all elements are
  // processed
  int mirror_idx = idx;
  if (mirror_idx < overlap / 2) {
    var_t x1, x2;
    var_t *xp1 = d_out + overlap - 1 - mirror_idx;
    var_t *yp1 = d_out + mirror_idx;
    const var_t *wp1 = window + mirror_idx;
    const var_t *wp2 = window + overlap - 1 - mirror_idx;

    x1 = *xp1;
    x2 = *yp1;

    // Use temporary variables to avoid writing order issues
    var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
    var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);

    *yp1 = temp1;
    *xp1 = temp2;
  }
}

__global__ void postAndMirrorKernelFused(var_t *d_out_ch0, var_t *d_out_ch1,
                                         const var_t *t, const var_t *window,
                                         int N2, int N4, int shift, var_t sine,
                                         int overlap) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Handle post-rotation part for both channels
  if (idx < (N4 + 1) >> 1) {
    // Channel 0 processing
    {
      var_t re, im, yr, yi;
      var_t t0, t1;

      // Calculate left pointer position for channel 0
      var_t *yp0 = d_out_ch0 + (overlap >> 1) + 2 * idx;
      // Calculate right pointer position for channel 0
      var_t *yp1 = d_out_ch0 + (overlap >> 1) + N2 - 2 - 2 * idx;

      // Process the first pair of values
      re = yp0[0];
      im = yp0[1];
      t0 = t[idx << shift];
      t1 = t[(N4 - idx) << shift];
      yr = S_MUL(re, t0) - S_MUL(im, t1);
      yi = S_MUL(im, t0) + S_MUL(re, t1);

      // Save the first pair of results
      var_t yr1 = yr;
      var_t yi1 = yi;

      // Process the second pair of values
      re = yp1[0];
      im = yp1[1];
      t0 = t[(N4 - idx - 1) << shift];
      t1 = t[(idx + 1) << shift];
      yr = S_MUL(re, t0) - S_MUL(im, t1);
      yi = S_MUL(im, t0) + S_MUL(re, t1);

      // Write results in the same order as the CPU version
      yp0[0] = -(yr1 - S_MUL(yi1, sine)); // Left real
      yp1[1] = yi1 + S_MUL(yr1, sine);    // Right imag
      yp1[0] = -(yr - S_MUL(yi, sine));   // Right real
      yp0[1] = yi + S_MUL(yr, sine);      // Left imag
    }

    // Channel 1 processing
    {
      var_t re, im, yr, yi;
      var_t t0, t1;

      // Calculate left pointer position for channel 1
      var_t *yp0 = d_out_ch1 + (overlap >> 1) + 2 * idx;
      // Calculate right pointer position for channel 1
      var_t *yp1 = d_out_ch1 + (overlap >> 1) + N2 - 2 - 2 * idx;

      // Process the first pair of values
      re = yp0[0];
      im = yp0[1];
      t0 = t[idx << shift];
      t1 = t[(N4 - idx) << shift];
      yr = S_MUL(re, t0) - S_MUL(im, t1);
      yi = S_MUL(im, t0) + S_MUL(re, t1);

      // Save the first pair of results
      var_t yr1 = yr;
      var_t yi1 = yi;

      // Process the second pair of values
      re = yp1[0];
      im = yp1[1];
      t0 = t[(N4 - idx - 1) << shift];
      t1 = t[(idx + 1) << shift];
      yr = S_MUL(re, t0) - S_MUL(im, t1);
      yi = S_MUL(im, t0) + S_MUL(re, t1);

      // Write results in the same order as the CPU version
      yp0[0] = -(yr1 - S_MUL(yi1, sine)); // Left real
      yp1[1] = yi1 + S_MUL(yr1, sine);    // Right imag
      yp1[0] = -(yr - S_MUL(yi, sine));   // Right real
      yp0[1] = yi + S_MUL(yr, sine);      // Left imag
    }
  }

  // sync threads before mirror operation
  __syncthreads();

  // Handle mirror part for both channels
  int mirror_idx = idx;
  if (mirror_idx < overlap / 2) {
    // Channel 0 mirror
    {
      var_t x1, x2;
      var_t *xp1 = d_out_ch0 + overlap - 1 - mirror_idx;
      var_t *yp1 = d_out_ch0 + mirror_idx;
      const var_t *wp1 = window + mirror_idx;
      const var_t *wp2 = window + overlap - 1 - mirror_idx;

      x1 = *xp1;
      x2 = *yp1;

      // Use temporary variables to avoid writing order issues
      var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
      var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);

      *yp1 = temp1;
      *xp1 = temp2;
    }

    // Channel 1 mirror
    {
      var_t x1, x2;
      var_t *xp1 = d_out_ch1 + overlap - 1 - mirror_idx;
      var_t *yp1 = d_out_ch1 + mirror_idx;
      const var_t *wp1 = window + mirror_idx;
      const var_t *wp2 = window + overlap - 1 - mirror_idx;

      x1 = *xp1;
      x2 = *yp1;

      // Use temporary variables to avoid writing order issues
      var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
      var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);

      *yp1 = temp1;
      *xp1 = temp2;
    }
  }
}

__global__ void doPreRotationFused(const var_t *xp1_ch0, const var_t *xp1_ch1,
                                   var_t *yp_ch0, var_t *yp_ch1, const var_t *t,
                                   int N4, int shift, int stride, int N2,
                                   var_t sine) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N4) {
    // Process channel 0
    const var_t *xp1_i_ch0 = xp1_ch0 + i * 2 * stride;
    const var_t *xp2_i_ch0 = xp1_ch0 + stride * (N2 - 1) - i * 2 * stride;

    var_t yr_ch0, yi_ch0;
    yr_ch0 = -S_MUL(*xp2_i_ch0, t[i << shift]) +
             S_MUL(*xp1_i_ch0, t[(N4 - i) << shift]);
    yi_ch0 = -S_MUL(*xp2_i_ch0, t[(N4 - i) << shift]) -
             S_MUL(*xp1_i_ch0, t[i << shift]);

    yp_ch0[i * 2] = yr_ch0 - S_MUL(yi_ch0, sine);
    yp_ch0[i * 2 + 1] = yi_ch0 + S_MUL(yr_ch0, sine);

    // Process channel 1
    const var_t *xp1_i_ch1 = xp1_ch1 + i * 2 * stride;
    const var_t *xp2_i_ch1 = xp1_ch1 + stride * (N2 - 1) - i * 2 * stride;

    var_t yr_ch1, yi_ch1;
    yr_ch1 = -S_MUL(*xp2_i_ch1, t[i << shift]) +
             S_MUL(*xp1_i_ch1, t[(N4 - i) << shift]);
    yi_ch1 = -S_MUL(*xp2_i_ch1, t[(N4 - i) << shift]) -
             S_MUL(*xp1_i_ch1, t[i << shift]);

    yp_ch1[i * 2] = yr_ch1 - S_MUL(yi_ch1, sine);
    yp_ch1[i * 2 + 1] = yi_ch1 + S_MUL(yr_ch1, sine);
  }
}

void processMDCTCuda(const var_t *input, var_t *output, const var_t *trig,
                     int N, int shift, int stride, var_t sine, int overlap,
                     const var_t *window) {
  int N2 = N >> 1;
  int N4 = N >> 2;

  // Device pointers and memory allocation
  var_t *dev_input, *dev_output, *dev_t, *dev_window, *dev_f2;
  size_t size_input = N4 * 2 * stride * sizeof(var_t);
  size_t size_output = (N2 + overlap) * sizeof(var_t);
  size_t size_fft = N4 * 2 * sizeof(var_t);
  size_t size_trig = (N4 << shift) * sizeof(var_t);
  size_t size_window = overlap * sizeof(var_t);

  // Allocate and copy memory
  CHECK_CUDA_ERROR(hipMalloc((void **)&dev_input, size_input));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dev_output, size_output));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dev_t, size_trig));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dev_window, size_window));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dev_f2, size_fft));

  CHECK_CUDA_ERROR(
      hipMemcpy(dev_output, output, size_output, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dev_input, input, size_input, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(dev_t, trig, size_trig, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dev_window, window, size_window, hipMemcpyHostToDevice));

  // Pre-rotation
  int blockSize = 256;
  int numBlocks = (N4 + blockSize - 1) / blockSize;
  doPreRotation<<<numBlocks, blockSize>>>(dev_input, dev_f2, dev_t, N4, shift,
                                          stride, N2, sine);
  hipDeviceSynchronize();

  // ifft
  cuda_fft_state *state = cuda_fft_alloc(N4, shift);
  if (!state) {
    fprintf(stderr, "Failed to allocate FFT state\n");
    exit(EXIT_FAILURE);
  }

  var_t *output_offset = dev_output + (overlap >> 1);
  int result =
      cuda_fft_execute(state, (const float *)dev_f2, (const float *)dev_f2,
                       (float *)output_offset, (float *)output_offset);

  if (result != 0) {
    fprintf(stderr, "FFT execution failed with error %d\n", result);
    cuda_fft_free(state);
    exit(EXIT_FAILURE);
  }

  CHECK_LAST_CUDA_ERROR(); // Check for errors after FFT execution
  hipDeviceSynchronize(); // Ensure all operations are complete

  // Fused post-rotation and mirror kernel
  // Calculate the maximum number of threads needed
  int max_elements = max((N4 + 1) >> 1, overlap / 2);
  int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
  postAndMirrorKernel<<<numBlocksFused, blockSize>>>(
      dev_output, dev_t, dev_window, N2, N4, shift, sine, overlap);
  CHECK_LAST_CUDA_ERROR();
  hipDeviceSynchronize();

  // Copy final results
  CHECK_CUDA_ERROR(
      hipMemcpy(output, dev_output, size_output, hipMemcpyDeviceToHost));

  // Cleanup
  if (state)
    cuda_fft_free(state);
  hipFree(dev_input);
  hipFree(dev_output);
  hipFree(dev_t);
  hipFree(dev_window);
  hipFree(dev_f2);
}

void printCudaVersion() {
  fprintf(stderr, "CUDA Compiled version: %d\n", __CUDACC_VER_MAJOR__);

  int runtime_ver;
  hipRuntimeGetVersion(&runtime_ver);
  fprintf(stderr, "CUDA Runtime version: %d\n", runtime_ver);

  int driver_ver;
  hipDriverGetVersion(&driver_ver);
  fprintf(stderr, "CUDA Driver version: %d\n", driver_ver);
}

#include <unordered_map>
static std::unordered_map<int, var_t *> dev_buf;
static std::unordered_map<int, cuda_fft_state *> fft_buf;

// Create MDCT CUDA state
mdct_cuda_state *mdct_cuda_create(int N, int shift, int stride, int overlap) {
  mdct_cuda_state *state = (mdct_cuda_state *)malloc(sizeof(mdct_cuda_state));
  if (!state)
    return nullptr;

  // Initialize configuration
  state->N = N;
  state->N2 = N >> 1;
  state->N4 = N >> 2;
  state->shift = shift;
  state->stride = stride;
  state->overlap = overlap;
  state->initialized = false;

  // Calculate buffer sizes
  state->size_input = state->N4 * 2 * stride * sizeof(var_t);
  state->size_output = (state->N2 + overlap) * sizeof(var_t);
  state->size_fft = state->N4 * 2 * sizeof(var_t);
  state->size_trig = (state->N4 << shift) * sizeof(var_t);
  state->size_window = overlap * sizeof(var_t);

  // Allocate device memory
  size_t total_size = state->size_input * 2 + state->size_output * 2 +
                      state->size_trig + state->size_window +
                      state->size_fft * 4;

  var_t *dev_buf;
  if (hipMalloc(&dev_buf, total_size) != hipSuccess) {
    free(state);
    return nullptr;
  }

  // Assign buffer pointers
  state->dev_input = dev_buf;
  state->dev_output = (var_t *)((char *)state->dev_input + state->size_input);
  state->dev_input1 = (var_t *)((char *)state->dev_output + state->size_output);
  state->dev_output1 = (var_t *)((char *)state->dev_input1 + state->size_input);
  state->dev_t = (var_t *)((char *)state->dev_output1 + state->size_output);
  state->dev_window = (var_t *)((char *)state->dev_t + state->size_trig);
  state->dev_f0 = (var_t *)((char *)state->dev_window + state->size_window);
  state->dev_f1 = (var_t *)((char *)state->dev_f0 + state->size_fft);
  state->dev_fft_output = (var_t *)((char *)state->dev_f1 + state->size_fft);

  // Create FFT plan
  if (hipfftPlan1d(&state->plan, state->N4, HIPFFT_C2C, 2) != HIPFFT_SUCCESS) {
    hipFree(dev_buf);
    free(state);
    return nullptr;
  }

  state->initialized = true;
  return state;
}

// Destroy MDCT CUDA state
void mdct_cuda_destroy(mdct_cuda_state *state) {
  if (state) {
    if (state->initialized) {
      hipFree(
          state->dev_input); // Free all device memory (allocated as one block)
      hipfftDestroy(state->plan);
    }
    free(state);
  }
}

// Process MDCT using persistent state
void mdct_cuda_process(mdct_cuda_state *state, const var_t *input[2],
                       var_t *output[2], const var_t *trig, const var_t *window,
                       var_t sine) {
  if (!state || !state->initialized)
    return;

  // Copy input data to device
  hipMemcpy(state->dev_input, input[0], state->size_input,
             hipMemcpyHostToDevice);
  hipMemcpy(state->dev_input1, input[1], state->size_input,
             hipMemcpyHostToDevice);
  hipMemcpy(state->dev_output, output[0], state->size_output,
             hipMemcpyHostToDevice);
  hipMemcpy(state->dev_output1, output[1], state->size_output,
             hipMemcpyHostToDevice);
  hipMemcpy(state->dev_t, trig, state->size_trig, hipMemcpyHostToDevice);
  hipMemcpy(state->dev_window, window, state->size_window,
             hipMemcpyHostToDevice);

  // Pre-rotation
  int blockSize = 256;
  int numBlocks = (state->N4 + blockSize - 1) / blockSize;

  doPreRotationFused<<<numBlocks, blockSize>>>(
      state->dev_input, state->dev_input1, state->dev_f0, state->dev_f1,
      state->dev_t, state->N4, state->shift, state->stride, state->N2, sine);

  // Execute FFT
  cuda_fft_state *state_fft = cuda_fft_alloc(state->N4, state->shift);
  if (!state_fft) {
    fprintf(stderr, "Failed to allocate FFT state\n");
    exit(EXIT_FAILURE);
  }

  var_t *c0_output_offset = state->dev_output + (state->overlap >> 1);
  var_t *c1_output_offset = state->dev_output1 + (state->overlap >> 1);
  int result = cuda_fft_execute(
      state_fft, (const float *)state->dev_f0, (const float *)state->dev_f1,
      (float *)c0_output_offset, (float *)c1_output_offset);

  if (result != 0) {
    fprintf(stderr, "FFT execution failed with error %d\n", result);
    cuda_fft_free(state_fft);
    exit(EXIT_FAILURE);
  }

  // Post-rotation and mirror
  int max_elements = max((state->N4 + 1) >> 1, state->overlap / 2);
  int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
  postAndMirrorKernelFused<<<numBlocksFused, blockSize>>>(
      state->dev_output, state->dev_output1, state->dev_t, state->dev_window,
      state->N2, state->N4, state->shift, sine, state->overlap);

  // Copy results back to host
  hipMemcpy(output[0], state->dev_output, state->size_output,
             hipMemcpyDeviceToHost);
  hipMemcpy(output[1], state->dev_output1, state->size_output,
             hipMemcpyDeviceToHost);
}

// Update the original function to use the new state management
void processMDCTCudaB1C2(const var_t *input[2], var_t *output[2],
                         const var_t *trig, int N, int shift, int stride,
                         var_t sine, int overlap, const var_t *window) {
  static mdct_cuda_state *state = nullptr;

  // Create state if not exists
  if (!state) {
    state = mdct_cuda_create(N, shift, stride, overlap);
    if (!state) {
      printf("Failed to create MDCT CUDA state\n");
      return;
    }
  }

  // Process using persistent state
  mdct_cuda_process(state, input, output, trig, window, sine);
}

// Update cleanup function
void cleanupCudaBuffers() {
  // Add cleanup for static state if needed
  // Note: This function might need to be called explicitly at program end
  for (auto &it : dev_buf) {
    hipFree(it.second);
  }
  dev_buf.clear();
  for (auto &it : fft_buf) {
    cuda_fft_free(it.second);
  }
  fft_buf.clear();
}

// Performance test function
void performanceTest(int numIterations) {
  // Test parameters
  const int N = 2048; // FFT size
  const int shift = 1;
  const int stride = 1;
  const float sine = 0.0f;
  const int overlap = 256;

  // Allocate host memory
  const int N2 = N >> 1;
  const int N4 = N >> 2;
  const var_t *input_const[2];
  var_t *input[2], *output[2], *trig, *window;

  input[0] = new var_t[N4 * 2 * stride];
  input[1] = new var_t[N4 * 2 * stride];
  output[0] = new var_t[N2 + overlap];
  output[1] = new var_t[N2 + overlap];
  trig = new var_t[N4 << shift];
  window = new var_t[overlap];

  input_const[0] = input[0];
  input_const[1] = input[1];

  // Initialize test data
  for (int i = 0; i < N4 * 2 * stride; i++) {
    input[0][i] = (var_t)rand() / RAND_MAX;
    input[1][i] = (var_t)rand() / RAND_MAX;
  }
  for (int i = 0; i < N4 << shift; i++) {
    trig[i] = (var_t)rand() / RAND_MAX;
  }
  for (int i = 0; i < overlap; i++) {
    window[i] = (var_t)rand() / RAND_MAX;
  }

  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Warmup run
  processMDCTCudaB1C2(input_const, output, trig, N, shift, stride, sine,
                      overlap, window);
  hipDeviceSynchronize();

  // Performance test
  float totalTime = 0.0f;
  float minTime = FLT_MAX;
  float maxTime = 0.0f;

  printf("\nRunning performance test with %d iterations...\n", numIterations);

  for (int i = 0; i < numIterations; i++) {
    hipEventRecord(start);

    processMDCTCudaB1C2(input_const, output, trig, N, shift, stride, sine,
                        overlap, window);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    totalTime += milliseconds;
    minTime = min(minTime, milliseconds);
    maxTime = max(maxTime, milliseconds);

    if ((i + 1) % 10 == 0) {
      printf("Completed %d iterations...\n", i + 1);
    }
  }

  // Print performance statistics
  float avgTime = totalTime / numIterations;
  printf("\nPerformance Statistics (over %d iterations):\n", numIterations);
  printf("Average Time: %.4f ms\n", avgTime);
  printf("Min Time:     %.4f ms\n", minTime);
  printf("Max Time:     %.4f ms\n", maxTime);
  printf("Total Time:   %.4f ms\n", totalTime);

  // Cleanup
  hipEventDestroy(start);
  hipEventDestroy(stop);

  delete[] input[0];
  delete[] input[1];
  delete[] output[0];
  delete[] output[1];
  delete[] trig;
  delete[] window;
}
