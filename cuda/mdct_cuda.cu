#include "hip/hip_runtime.h"
#include "mdct_cuda.hpp"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#ifndef S_MUL
#define S_MUL(a, b) ((a) * (b))
#endif

#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define CHECK_LAST_CUDA_ERROR() do { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// static float total_kernel_time = 0.0f;
// static int call_count = 0;

// CUDA kernel
__global__ void doPreRotation(const var_t *xp1, var_t *yp, const var_t *t,
                              int N4, int shift, int stride, int N2,
                              var_t sine) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N4) {
        const var_t *xp1_i = xp1 + i * 2 * stride;
        const var_t *xp2_i = xp1 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr, yi;
        yr = -S_MUL(*xp2_i, t[i << shift]) + S_MUL(*xp1_i, t[(N4 - i) << shift]);
        yi = -S_MUL(*xp2_i, t[(N4 - i) << shift]) - S_MUL(*xp1_i, t[i << shift]);

        yp[i * 2] = yr - S_MUL(yi, sine);
        yp[i * 2 + 1] = yi + S_MUL(yr, sine);
    }
}

void preRotateWithCuda(const var_t *host_xp1, var_t *host_yp,
                       const var_t *host_t, int N, int shift, int stride,
                       var_t sine) {
    int N2 = N >> 1;
    int N4 = N >> 2;
    var_t *dev_xp1;
    var_t *dev_yp;
    var_t *dev_t;

    hipMalloc((void **)&dev_xp1, N4 * 2 * stride * sizeof(var_t));
    hipMalloc((void **)&dev_yp, N4 * 2 * sizeof(var_t));
    hipMalloc((void **)&dev_t, (N4 << shift) * sizeof(var_t));

    hipMemcpy(dev_xp1, host_xp1, N4 * 2 * stride * sizeof(var_t),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_t, host_t, (N4 << shift) * sizeof(var_t),
               hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;

    doPreRotation<<<numBlocks, blockSize>>>(dev_xp1, dev_yp, dev_t, N4, shift,
                                          stride, N2, sine);

    hipDeviceSynchronize();
    hipMemcpy(host_yp, dev_yp, N4 * 2 * sizeof(var_t), hipMemcpyDeviceToHost);

    hipFree(dev_xp1);
    hipFree(dev_yp);
    hipFree(dev_t);
}

__global__ void postAndMirrorKernel(var_t *d_out, 
                                   const var_t *t, 
                                   const var_t *window,
                                   int N2, int N4, 
                                   int shift,
                                   var_t sine, 
                                   int overlap) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle post-rotation part
    if (idx < (N4 + 1) >> 1) {
        var_t re, im, yr, yi;
        var_t t0, t1;
        
        // Calculate left pointer position
        var_t *yp0 = d_out + (overlap >> 1) + 2 * idx;
        // Calculate right pointer position
        var_t *yp1 = d_out + (overlap >> 1) + N2 - 2 - 2 * idx;
        
        // Process the first pair of values
        re = yp0[0];
        im = yp0[1];
        t0 = t[idx << shift];
        t1 = t[(N4 - idx) << shift];
        yr = S_MUL(re, t0) - S_MUL(im, t1);
        yi = S_MUL(im, t0) + S_MUL(re, t1);
        
        // Save the first pair of results
        var_t yr1 = yr;
        var_t yi1 = yi;
        
        // Process the second pair of values
        re = yp1[0];
        im = yp1[1];
        t0 = t[(N4 - idx - 1) << shift];
        t1 = t[(idx + 1) << shift];
        yr = S_MUL(re, t0) - S_MUL(im, t1);
        yi = S_MUL(im, t0) + S_MUL(re, t1);
        
        // Write results in the same order as the CPU version
        yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
        yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
        yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
        yp0[1] = yi + S_MUL(yr, sine);       // Left imag
    }


    //sync
    __syncthreads();

    // Handle mirror part
    // Use a different index for the mirror operation to ensure all elements are processed
    int mirror_idx = idx;
    if (mirror_idx < overlap / 2) {
        var_t x1, x2;
        var_t *xp1 = d_out + overlap - 1 - mirror_idx;
        var_t *yp1 = d_out + mirror_idx;
        const var_t *wp1 = window + mirror_idx;
        const var_t *wp2 = window + overlap - 1 - mirror_idx;
        
        x1 = *xp1;
        x2 = *yp1;
        
        // Use temporary variables to avoid writing order issues
        var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
        var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
        
        *yp1 = temp1;
        *xp1 = temp2;
    }
}

__global__ void postAndMirrorKernelFused(var_t *d_out_ch0, 
                                        var_t *d_out_ch1,
                                        const var_t *t, 
                                        const var_t *window,
                                        int N2, int N4, 
                                        int shift,
                                        var_t sine, 
                                        int overlap) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle post-rotation part for both channels
    if (idx < (N4 + 1) >> 1) {
        // Channel 0 processing
        {
            var_t re, im, yr, yi;
            var_t t0, t1;
            
            // Calculate left pointer position for channel 0
            var_t *yp0 = d_out_ch0 + (overlap >> 1) + 2 * idx;
            // Calculate right pointer position for channel 0
            var_t *yp1 = d_out_ch0 + (overlap >> 1) + N2 - 2 - 2 * idx;
            
            // Process the first pair of values
            re = yp0[0];
            im = yp0[1];
            t0 = t[idx << shift];
            t1 = t[(N4 - idx) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Save the first pair of results
            var_t yr1 = yr;
            var_t yi1 = yi;
            
            // Process the second pair of values
            re = yp1[0];
            im = yp1[1];
            t0 = t[(N4 - idx - 1) << shift];
            t1 = t[(idx + 1) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Write results in the same order as the CPU version
            yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
            yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
            yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
            yp0[1] = yi + S_MUL(yr, sine);       // Left imag
        }

        // Channel 1 processing
        {
            var_t re, im, yr, yi;
            var_t t0, t1;
            
            // Calculate left pointer position for channel 1
            var_t *yp0 = d_out_ch1 + (overlap >> 1) + 2 * idx;
            // Calculate right pointer position for channel 1
            var_t *yp1 = d_out_ch1 + (overlap >> 1) + N2 - 2 - 2 * idx;
            
            // Process the first pair of values
            re = yp0[0];
            im = yp0[1];
            t0 = t[idx << shift];
            t1 = t[(N4 - idx) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Save the first pair of results
            var_t yr1 = yr;
            var_t yi1 = yi;
            
            // Process the second pair of values
            re = yp1[0];
            im = yp1[1];
            t0 = t[(N4 - idx - 1) << shift];
            t1 = t[(idx + 1) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Write results in the same order as the CPU version
            yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
            yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
            yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
            yp0[1] = yi + S_MUL(yr, sine);       // Left imag
        }
    }

    //sync threads before mirror operation
    __syncthreads();

    // Handle mirror part for both channels
    int mirror_idx = idx;
    if (mirror_idx < overlap / 2) {
        // Channel 0 mirror
        {
            var_t x1, x2;
            var_t *xp1 = d_out_ch0 + overlap - 1 - mirror_idx;
            var_t *yp1 = d_out_ch0 + mirror_idx;
            const var_t *wp1 = window + mirror_idx;
            const var_t *wp2 = window + overlap - 1 - mirror_idx;
            
            x1 = *xp1;
            x2 = *yp1;
            
            // Use temporary variables to avoid writing order issues
            var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
            var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
            
            *yp1 = temp1;
            *xp1 = temp2;
        }

        // Channel 1 mirror
        {
            var_t x1, x2;
            var_t *xp1 = d_out_ch1 + overlap - 1 - mirror_idx;
            var_t *yp1 = d_out_ch1 + mirror_idx;
            const var_t *wp1 = window + mirror_idx;
            const var_t *wp2 = window + overlap - 1 - mirror_idx;
            
            x1 = *xp1;
            x2 = *yp1;
            
            // Use temporary variables to avoid writing order issues
            var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
            var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
            
            *yp1 = temp1;
            *xp1 = temp2;
        }
    }
}

__global__ void doPreRotationFused(const var_t *xp1_ch0, const var_t *xp1_ch1,
                                  var_t *yp_ch0, var_t *yp_ch1,
                                  const var_t *t, int N4, int shift,
                                  int stride, int N2, var_t sine) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N4) {
        // Process channel 0
        const var_t *xp1_i_ch0 = xp1_ch0 + i * 2 * stride;
        const var_t *xp2_i_ch0 = xp1_ch0 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr_ch0, yi_ch0;
        yr_ch0 = -S_MUL(*xp2_i_ch0, t[i << shift]) + S_MUL(*xp1_i_ch0, t[(N4 - i) << shift]);
        yi_ch0 = -S_MUL(*xp2_i_ch0, t[(N4 - i) << shift]) - S_MUL(*xp1_i_ch0, t[i << shift]);

        yp_ch0[i * 2] = yr_ch0 - S_MUL(yi_ch0, sine);
        yp_ch0[i * 2 + 1] = yi_ch0 + S_MUL(yr_ch0, sine);

        // Process channel 1
        const var_t *xp1_i_ch1 = xp1_ch1 + i * 2 * stride;
        const var_t *xp2_i_ch1 = xp1_ch1 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr_ch1, yi_ch1;
        yr_ch1 = -S_MUL(*xp2_i_ch1, t[i << shift]) + S_MUL(*xp1_i_ch1, t[(N4 - i) << shift]);
        yi_ch1 = -S_MUL(*xp2_i_ch1, t[(N4 - i) << shift]) - S_MUL(*xp1_i_ch1, t[i << shift]);

        yp_ch1[i * 2] = yr_ch1 - S_MUL(yi_ch1, sine);
        yp_ch1[i * 2 + 1] = yi_ch1 + S_MUL(yr_ch1, sine);
    }
}

void processMDCTCuda(const var_t *input, var_t *output, const var_t *trig, int N, 
                     int shift, int stride, var_t sine, int overlap, const var_t *window) {
    int N2 = N >> 1;
    int N4 = N >> 2;

    // Device pointers and memory allocation
    var_t *dev_input, *dev_output, *dev_t, *dev_window, *dev_f2;
    size_t size_input = N4 * 2 * stride * sizeof(var_t);
    size_t size_output = (N2 + overlap) * sizeof(var_t);
    size_t size_fft = N4 * 2 * sizeof(var_t);
    size_t size_trig = (N4 << shift) * sizeof(var_t);
    size_t size_window = overlap * sizeof(var_t);

    // Allocate and copy memory
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_input, size_input));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_output, size_output));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_t, size_trig));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_window, size_window));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_f2, size_fft));

    CHECK_CUDA_ERROR(hipMemcpy(dev_output, output, size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input, input, size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_t, trig, size_trig, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_window, window, size_window, hipMemcpyHostToDevice));

    // Pre-rotation
    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;
    doPreRotation<<<numBlocks, blockSize>>>(dev_input, dev_f2, dev_t, N4, shift, stride, N2, sine);
    hipDeviceSynchronize();
    
    // ifft
    cuda_fft_state *state = cuda_fft_alloc(N4, shift);
    if (!state) {
        fprintf(stderr, "Failed to allocate FFT state\n");
        exit(EXIT_FAILURE);
    }

    var_t *output_offset = dev_output + (overlap >> 1);
    hipfftResult result = hipfftExecC2C(state->plan,
                                      (hipfftComplex *)dev_f2,
                                      (hipfftComplex *)output_offset,
                                      HIPFFT_BACKWARD);
    CHECK_LAST_CUDA_ERROR(); // Check for errors after FFT execution
    hipDeviceSynchronize(); // Ensure all operations are complete

    // Fused post-rotation and mirror kernel
    // Calculate the maximum number of threads needed
    int max_elements = max((N4 + 1) >> 1, overlap / 2);
    int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
    postAndMirrorKernel<<<numBlocksFused, blockSize>>>(dev_output, dev_t, dev_window,
                                                      N2, N4, shift, sine, overlap);
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();

    // Copy final results
    CHECK_CUDA_ERROR(hipMemcpy(output, dev_output, size_output, hipMemcpyDeviceToHost));

    // Cleanup
    if (state) cuda_fft_free(state);
    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_t);
    hipFree(dev_window);
    hipFree(dev_f2);
}


void printCudaVersion() {
    fprintf(stderr, "CUDA Compiled version: %d\n", __CUDACC_VER_MAJOR__);

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    fprintf(stderr, "CUDA Runtime version: %d\n", runtime_ver);

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    fprintf(stderr, "CUDA Driver version: %d\n", driver_ver);
}

#include <unordered_map>
static std::unordered_map<int, var_t *> dev_buf;
static std::unordered_map<int, cuda_fft_state *> fft_buf;

void processMDCTCudaB1C2(const var_t **input, var_t **output, const var_t *trig, int N,
                         int shift, int stride, var_t sine, int overlap, const var_t *window)
{
    int N2 = N >> 1;
    int N4 = N >> 2;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEvent_t event1, event2, event3, event4, event5, event6, event7;
    float total_time = 0.0f, time_temp = 0.0f;
    float mem_alloc_time = 0.0f, h2d_time = 0.0f, preproc_time = 0.0f;
    float ifft_plan_time = 0.0f, ifft_exec_time = 0.0f, ifft_cleanup_time = 0.0f;
    float postproc_time = 0.0f, d2h_time = 0.0f, cleanup_time = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventCreate(&event3);
    hipEventCreate(&event4);
    hipEventCreate(&event5);
    hipEventCreate(&event6);
    hipEventCreate(&event7);

    hipEventRecord(start);

    // Device pointers and memory allocation
    var_t *dev_input, *dev_output, *dev_t, *dev_window, *dev_f0, *dev_f1;
    var_t *dev_input1, *dev_output1;
    size_t size_input = N4 * 2 * stride * sizeof(var_t);
    size_t size_output = (N2 + overlap) * sizeof(var_t);
    size_t size_fft = N4 * 2 * sizeof(var_t);
    size_t size_trig = (N4 << shift) * sizeof(var_t);
    size_t size_window = overlap * sizeof(var_t);

    // Allocate memory
    size_t total_dev_size = size_input * 2 + size_output * 2 + size_trig + size_window + size_fft * 4;
    var_t *dev_buf_ptr;
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_buf_ptr, total_dev_size));
    dev_input = dev_buf_ptr;
    dev_output = (float*)((char *)dev_input + size_input);
    dev_input1 = (float*)((char *)dev_output + size_output);
    dev_output1 = (float*)((char *)dev_input1 + size_input);
    dev_t = (float*)((char *)dev_output1 + size_output);
    dev_window = (float*)((char *)dev_t + size_trig);
    dev_f0 = (float*)((char *)dev_window + size_window);
    dev_f1 = (float*)((char *)dev_f0 + size_fft);
    var_t *dev_fft_output = (float*)((char *)dev_f1 + size_fft);

    hipEventRecord(event1);

    // Host to Device transfers
    CHECK_CUDA_ERROR(hipMemcpy(dev_output, output[0], size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input, input[0], size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_output1, output[1], size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input1, input[1], size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_t, trig, size_trig, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_window, window, size_window, hipMemcpyHostToDevice));

    hipEventRecord(event2);

    // Pre-rotation
    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;
    doPreRotationFused<<<numBlocks, blockSize>>>(dev_input, dev_input1, dev_f0, dev_f1, dev_t, N4, shift, stride, N2, sine);
    hipDeviceSynchronize();

    hipEventRecord(event3);

    // IFFT Plan
    hipfftHandle plan;
    hipfftResult result = hipfftPlan1d(&plan, N4, HIPFFT_C2C, 2);
    if (result != HIPFFT_SUCCESS)
    {
        exit(EXIT_FAILURE);
    }

    hipEventRecord(event4);

    // IFFT Execute
    result = hipfftExecC2C(plan,
                          (hipfftComplex *)dev_f0,
                          (hipfftComplex *)dev_fft_output,
                          HIPFFT_BACKWARD);
    hipDeviceSynchronize();

    hipEventRecord(event5);

    // IFFT Cleanup
    hipfftDestroy(plan);

    hipEventRecord(event6);

    // Post-processing
    var_t *c0_output_offset = dev_output + (overlap >> 1);
    var_t *c1_output_offset = dev_output1 + (overlap >> 1);
    CHECK_CUDA_ERROR(hipMemcpy(c0_output_offset, dev_fft_output, size_fft, hipMemcpyDeviceToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(c1_output_offset, (char *)dev_fft_output + size_fft, size_fft, hipMemcpyDeviceToDevice));
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();

    int max_elements = max((N4 + 1) >> 1, overlap / 2);
    int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
    postAndMirrorKernelFused<<<numBlocksFused, blockSize>>>(dev_output, dev_output1, dev_t, dev_window,
                                                           N2, N4, shift, sine, overlap);
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();

    hipEventRecord(event7);

    // Device to Host transfer
    CHECK_CUDA_ERROR(hipMemcpy(output[0], dev_output, size_output, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(output[1], dev_output1, size_output, hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate timing for each section
    hipEventElapsedTime(&mem_alloc_time, start, event1);
    hipEventElapsedTime(&h2d_time, event1, event2);
    hipEventElapsedTime(&preproc_time, event2, event3);
    hipEventElapsedTime(&ifft_plan_time, event3, event4);
    hipEventElapsedTime(&ifft_exec_time, event4, event5);
    hipEventElapsedTime(&ifft_cleanup_time, event5, event6);
    hipEventElapsedTime(&postproc_time, event6, event7);
    hipEventElapsedTime(&d2h_time, event7, stop);
    hipEventElapsedTime(&total_time, start, stop);

    // Calculate IFFT overhead
    float ifft_total = ifft_plan_time + ifft_exec_time + ifft_cleanup_time;
    float other_time = total_time - (mem_alloc_time + h2d_time + preproc_time + 
                                   ifft_total + postproc_time + d2h_time);

    // Print timing statistics
    printf("\nMDCT CUDA Timing Statistics:\n");
    printf("Total Time:                  %.3f ms (100.0%%)\n", total_time);
    printf("Memory Allocation:           %.3f ms (%5.1f%%)\n", mem_alloc_time, mem_alloc_time/total_time*100);
    printf("Host to Device Transfer:     %.3f ms (%5.1f%%)\n", h2d_time, h2d_time/total_time*100);
    printf("Pre-processing:              %.3f ms (%5.1f%%)\n", preproc_time, preproc_time/total_time*100);
    printf("IFFT Total:                  %.3f ms (%5.1f%%)\n", ifft_total, ifft_total/total_time*100);
    printf("  IFFT Plan:                 %.3f ms (%5.1f%%)\n", ifft_plan_time, ifft_plan_time/total_time*100);
    printf("  IFFT Execute:              %.3f ms (%5.1f%%)\n", ifft_exec_time, ifft_exec_time/total_time*100);
    printf("  IFFT Cleanup:              %.3f ms (%5.1f%%)\n", ifft_cleanup_time, ifft_cleanup_time/total_time*100);
    printf("Post-processing:             %.3f ms (%5.1f%%)\n", postproc_time, postproc_time/total_time*100);
    printf("Device to Host Transfer:     %.3f ms (%5.1f%%)\n", d2h_time, d2h_time/total_time*100);
    printf("Other/Overhead:              %.3f ms (%5.1f%%)\n", other_time, other_time/total_time*100);

    // Cleanup events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(event1);
    hipEventDestroy(event2);
    hipEventDestroy(event3);
    hipEventDestroy(event4);
    hipEventDestroy(event5);
    hipEventDestroy(event6);
    hipEventDestroy(event7);

    // Cleanup device memory
    hipFree(dev_buf_ptr);
}

void cleanupCudaBuffers() {
    for (auto &it : dev_buf) {
        hipFree(it.second);
    }
    dev_buf.clear();
    for (auto &it : fft_buf) {
        cuda_fft_free(it.second);
    }
    fft_buf.clear();
}
