#include "hip/hip_runtime.h"
#include "mdct_cuda.hpp"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

#ifndef S_MUL
#define S_MUL(a, b) ((a) * (b))
#endif

#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

#define CHECK_LAST_CUDA_ERROR() do { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

static float total_kernel_time = 0.0f;
static int call_count = 0;

// CUDA kernel
__global__ void doPreRotation(const var_t *xp1, var_t *yp, const var_t *t,
                              int N4, int shift, int stride, int N2,
                              var_t sine) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N4) {
        const var_t *xp1_i = xp1 + i * 2 * stride;
        const var_t *xp2_i = xp1 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr, yi;
        yr = -S_MUL(*xp2_i, t[i << shift]) + S_MUL(*xp1_i, t[(N4 - i) << shift]);
        yi = -S_MUL(*xp2_i, t[(N4 - i) << shift]) - S_MUL(*xp1_i, t[i << shift]);

        yp[i * 2] = yr - S_MUL(yi, sine);
        yp[i * 2 + 1] = yi + S_MUL(yr, sine);
    }
}

void preRotateWithCuda(const var_t *host_xp1, var_t *host_yp,
                       const var_t *host_t, int N, int shift, int stride,
                       var_t sine) {
    int N2 = N >> 1;
    int N4 = N >> 2;
    var_t *dev_xp1;
    var_t *dev_yp;
    var_t *dev_t;

    hipMalloc((void **)&dev_xp1, N4 * 2 * stride * sizeof(var_t));
    hipMalloc((void **)&dev_yp, N4 * 2 * sizeof(var_t));
    hipMalloc((void **)&dev_t, (N4 << shift) * sizeof(var_t));

    hipMemcpy(dev_xp1, host_xp1, N4 * 2 * stride * sizeof(var_t),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_t, host_t, (N4 << shift) * sizeof(var_t),
               hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;

    doPreRotation<<<numBlocks, blockSize>>>(dev_xp1, dev_yp, dev_t, N4, shift,
                                          stride, N2, sine);

    hipDeviceSynchronize();
    hipMemcpy(host_yp, dev_yp, N4 * 2 * sizeof(var_t), hipMemcpyDeviceToHost);

    hipFree(dev_xp1);
    hipFree(dev_yp);
    hipFree(dev_t);
}

__global__ void postAndMirrorKernel(var_t *d_out, 
                                   const var_t *t, 
                                   const var_t *window,
                                   int N2, int N4, 
                                   int shift,
                                   var_t sine, 
                                   int overlap) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle post-rotation part
    if (idx < (N4 + 1) >> 1) {
        var_t re, im, yr, yi;
        var_t t0, t1;
        
        // Calculate left pointer position
        var_t *yp0 = d_out + (overlap >> 1) + 2 * idx;
        // Calculate right pointer position
        var_t *yp1 = d_out + (overlap >> 1) + N2 - 2 - 2 * idx;
        
        // Process the first pair of values
        re = yp0[0];
        im = yp0[1];
        t0 = t[idx << shift];
        t1 = t[(N4 - idx) << shift];
        yr = S_MUL(re, t0) - S_MUL(im, t1);
        yi = S_MUL(im, t0) + S_MUL(re, t1);
        
        // Save the first pair of results
        var_t yr1 = yr;
        var_t yi1 = yi;
        
        // Process the second pair of values
        re = yp1[0];
        im = yp1[1];
        t0 = t[(N4 - idx - 1) << shift];
        t1 = t[(idx + 1) << shift];
        yr = S_MUL(re, t0) - S_MUL(im, t1);
        yi = S_MUL(im, t0) + S_MUL(re, t1);
        
        // Write results in the same order as the CPU version
        yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
        yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
        yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
        yp0[1] = yi + S_MUL(yr, sine);       // Left imag
    }


    //sync
    __syncthreads();

    // Handle mirror part
    // Use a different index for the mirror operation to ensure all elements are processed
    int mirror_idx = idx;
    if (mirror_idx < overlap / 2) {
        var_t x1, x2;
        var_t *xp1 = d_out + overlap - 1 - mirror_idx;
        var_t *yp1 = d_out + mirror_idx;
        const var_t *wp1 = window + mirror_idx;
        const var_t *wp2 = window + overlap - 1 - mirror_idx;
        
        x1 = *xp1;
        x2 = *yp1;
        
        // Use temporary variables to avoid writing order issues
        var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
        var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
        
        *yp1 = temp1;
        *xp1 = temp2;
    }
}

__global__ void postAndMirrorKernelFused(var_t *d_out_ch0, 
                                        var_t *d_out_ch1,
                                        const var_t *t, 
                                        const var_t *window,
                                        int N2, int N4, 
                                        int shift,
                                        var_t sine, 
                                        int overlap) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handle post-rotation part for both channels
    if (idx < (N4 + 1) >> 1) {
        // Channel 0 processing
        {
            var_t re, im, yr, yi;
            var_t t0, t1;
            
            // Calculate left pointer position for channel 0
            var_t *yp0 = d_out_ch0 + (overlap >> 1) + 2 * idx;
            // Calculate right pointer position for channel 0
            var_t *yp1 = d_out_ch0 + (overlap >> 1) + N2 - 2 - 2 * idx;
            
            // Process the first pair of values
            re = yp0[0];
            im = yp0[1];
            t0 = t[idx << shift];
            t1 = t[(N4 - idx) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Save the first pair of results
            var_t yr1 = yr;
            var_t yi1 = yi;
            
            // Process the second pair of values
            re = yp1[0];
            im = yp1[1];
            t0 = t[(N4 - idx - 1) << shift];
            t1 = t[(idx + 1) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Write results in the same order as the CPU version
            yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
            yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
            yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
            yp0[1] = yi + S_MUL(yr, sine);       // Left imag
        }

        // Channel 1 processing
        {
            var_t re, im, yr, yi;
            var_t t0, t1;
            
            // Calculate left pointer position for channel 1
            var_t *yp0 = d_out_ch1 + (overlap >> 1) + 2 * idx;
            // Calculate right pointer position for channel 1
            var_t *yp1 = d_out_ch1 + (overlap >> 1) + N2 - 2 - 2 * idx;
            
            // Process the first pair of values
            re = yp0[0];
            im = yp0[1];
            t0 = t[idx << shift];
            t1 = t[(N4 - idx) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Save the first pair of results
            var_t yr1 = yr;
            var_t yi1 = yi;
            
            // Process the second pair of values
            re = yp1[0];
            im = yp1[1];
            t0 = t[(N4 - idx - 1) << shift];
            t1 = t[(idx + 1) << shift];
            yr = S_MUL(re, t0) - S_MUL(im, t1);
            yi = S_MUL(im, t0) + S_MUL(re, t1);
            
            // Write results in the same order as the CPU version
            yp0[0] = -(yr1 - S_MUL(yi1, sine));  // Left real
            yp1[1] = yi1 + S_MUL(yr1, sine);     // Right imag
            yp1[0] = -(yr - S_MUL(yi, sine));    // Right real
            yp0[1] = yi + S_MUL(yr, sine);       // Left imag
        }
    }

    //sync threads before mirror operation
    __syncthreads();

    // Handle mirror part for both channels
    int mirror_idx = idx;
    if (mirror_idx < overlap / 2) {
        // Channel 0 mirror
        {
            var_t x1, x2;
            var_t *xp1 = d_out_ch0 + overlap - 1 - mirror_idx;
            var_t *yp1 = d_out_ch0 + mirror_idx;
            const var_t *wp1 = window + mirror_idx;
            const var_t *wp2 = window + overlap - 1 - mirror_idx;
            
            x1 = *xp1;
            x2 = *yp1;
            
            // Use temporary variables to avoid writing order issues
            var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
            var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
            
            *yp1 = temp1;
            *xp1 = temp2;
        }

        // Channel 1 mirror
        {
            var_t x1, x2;
            var_t *xp1 = d_out_ch1 + overlap - 1 - mirror_idx;
            var_t *yp1 = d_out_ch1 + mirror_idx;
            const var_t *wp1 = window + mirror_idx;
            const var_t *wp2 = window + overlap - 1 - mirror_idx;
            
            x1 = *xp1;
            x2 = *yp1;
            
            // Use temporary variables to avoid writing order issues
            var_t temp1 = S_MUL(*wp2, x2) - S_MUL(*wp1, x1);
            var_t temp2 = S_MUL(*wp1, x2) + S_MUL(*wp2, x1);
            
            *yp1 = temp1;
            *xp1 = temp2;
        }
    }
}

__global__ void doPreRotationFused(const var_t *xp1_ch0, const var_t *xp1_ch1,
                                  var_t *yp_ch0, var_t *yp_ch1,
                                  const var_t *t, int N4, int shift,
                                  int stride, int N2, var_t sine) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N4) {
        // Process channel 0
        const var_t *xp1_i_ch0 = xp1_ch0 + i * 2 * stride;
        const var_t *xp2_i_ch0 = xp1_ch0 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr_ch0, yi_ch0;
        yr_ch0 = -S_MUL(*xp2_i_ch0, t[i << shift]) + S_MUL(*xp1_i_ch0, t[(N4 - i) << shift]);
        yi_ch0 = -S_MUL(*xp2_i_ch0, t[(N4 - i) << shift]) - S_MUL(*xp1_i_ch0, t[i << shift]);

        yp_ch0[i * 2] = yr_ch0 - S_MUL(yi_ch0, sine);
        yp_ch0[i * 2 + 1] = yi_ch0 + S_MUL(yr_ch0, sine);

        // Process channel 1
        const var_t *xp1_i_ch1 = xp1_ch1 + i * 2 * stride;
        const var_t *xp2_i_ch1 = xp1_ch1 + stride * (N2 - 1) - i * 2 * stride;

        var_t yr_ch1, yi_ch1;
        yr_ch1 = -S_MUL(*xp2_i_ch1, t[i << shift]) + S_MUL(*xp1_i_ch1, t[(N4 - i) << shift]);
        yi_ch1 = -S_MUL(*xp2_i_ch1, t[(N4 - i) << shift]) - S_MUL(*xp1_i_ch1, t[i << shift]);

        yp_ch1[i * 2] = yr_ch1 - S_MUL(yi_ch1, sine);
        yp_ch1[i * 2 + 1] = yi_ch1 + S_MUL(yr_ch1, sine);
    }
}

void processMDCTCuda(const var_t *input, var_t *output, const var_t *trig, int N, 
                     int shift, int stride, var_t sine, int overlap, const var_t *window) {
    int N2 = N >> 1;
    int N4 = N >> 2;

    // Device pointers and memory allocation
    var_t *dev_input, *dev_output, *dev_t, *dev_window, *dev_f2;
    size_t size_input = N4 * 2 * stride * sizeof(var_t);
    size_t size_output = (N2 + overlap) * sizeof(var_t);
    size_t size_fft = N4 * 2 * sizeof(var_t);
    size_t size_trig = (N4 << shift) * sizeof(var_t);
    size_t size_window = overlap * sizeof(var_t);

    // Allocate and copy memory
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_input, size_input));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_output, size_output));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_t, size_trig));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_window, size_window));
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_f2, size_fft));

    CHECK_CUDA_ERROR(hipMemcpy(dev_output, output, size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input, input, size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_t, trig, size_trig, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_window, window, size_window, hipMemcpyHostToDevice));

    // Pre-rotation
    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;
    doPreRotation<<<numBlocks, blockSize>>>(dev_input, dev_f2, dev_t, N4, shift, stride, N2, sine);
    hipDeviceSynchronize();
    
    // ifft
    cuda_fft_state *state = cuda_fft_alloc(N4, shift);
    if (!state) {
        fprintf(stderr, "Failed to allocate FFT state\n");
        exit(EXIT_FAILURE);
    }

    var_t *output_offset = dev_output + (overlap >> 1);
    hipfftResult result = hipfftExecC2C(state->plan,
                                      (hipfftComplex *)dev_f2,
                                      (hipfftComplex *)output_offset,
                                      HIPFFT_BACKWARD);
    CHECK_LAST_CUDA_ERROR(); // Check for errors after FFT execution
    hipDeviceSynchronize(); // Ensure all operations are complete

    // Fused post-rotation and mirror kernel
    // Calculate the maximum number of threads needed
    int max_elements = max((N4 + 1) >> 1, overlap / 2);
    int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
    postAndMirrorKernel<<<numBlocksFused, blockSize>>>(dev_output, dev_t, dev_window,
                                                      N2, N4, shift, sine, overlap);
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();

    // Copy final results
    CHECK_CUDA_ERROR(hipMemcpy(output, dev_output, size_output, hipMemcpyDeviceToHost));

    // Cleanup
    if (state) cuda_fft_free(state);
    hipFree(dev_input);
    hipFree(dev_output);
    hipFree(dev_t);
    hipFree(dev_window);
    hipFree(dev_f2);
}


void printCudaVersion() {
    fprintf(stderr, "CUDA Compiled version: %d\n", __CUDACC_VER_MAJOR__);

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    fprintf(stderr, "CUDA Runtime version: %d\n", runtime_ver);

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    fprintf(stderr, "CUDA Driver version: %d\n", driver_ver);
}

#include <unordered_map>
static std::unordered_map<int, var_t *> dev_buf;
static std::unordered_map<int, cuda_fft_state *> fft_buf;

void processMDCTCudaB1C2(const var_t *input[2], var_t *output[2], const var_t *trig, int N,
                         int shift, int stride, var_t sine, int overlap, const var_t *window)
{
    int N2 = N >> 1;
    int N4 = N >> 2;

    // Device pointers and memory allocation
    var_t *dev_input, *dev_output, *dev_t, *dev_window, *dev_f0, *dev_f1;
    var_t *dev_input1, *dev_output1;
    size_t size_input = N4 * 2 * stride * sizeof(var_t);
    size_t size_output = (N2 + overlap) * sizeof(var_t);
    size_t size_fft = N4 * 2 * sizeof(var_t);
    size_t size_trig = (N4 << shift) * sizeof(var_t);
    size_t size_window = overlap * sizeof(var_t);

    // Allocate and copy memory
    size_t total_dev_size = size_input * 2 + size_output * 2 + size_trig + size_window + size_fft * 4;
    var_t *dev_buf_ptr;
    CHECK_CUDA_ERROR(hipMalloc((void **)&dev_buf_ptr, total_dev_size));
    dev_input = dev_buf_ptr;
    dev_output = (float*)((char *)dev_input + size_input);
    dev_input1 = (float*)((char *)dev_output + size_output);
    dev_output1 = (float*)((char *)dev_input1 + size_input);
    dev_t = (float*)((char *)dev_output1 + size_output);
    dev_window = (float*)((char *)dev_t + size_trig);
    dev_f0 = (float*)((char *)dev_window + size_window);
    dev_f1 = (float*)((char *)dev_f0 + size_fft);
    var_t *dev_fft_output = (float*)((char *)dev_f1 + size_fft);

    // if(dev_buf.find(total_dev_size) == dev_buf.end()) {
    //     CHECK_CUDA_ERROR(hipMalloc((void **)&dev_buf_ptr, total_dev_size));
    //     dev_buf[total_dev_size] = dev_buf_ptr;
    // } else {
    //     dev_buf_ptr = dev_buf[total_dev_size];
    // }

    // make sure to copy output to device !!!
    CHECK_CUDA_ERROR(hipMemcpy(dev_output, output[0], size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input, input[0], size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_output1, output[1], size_output, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_input1, input[1], size_input, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_t, trig, size_trig, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(dev_window, window, size_window, hipMemcpyHostToDevice));

    // Create CUDA events for GPU kernel timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Start GPU kernel timing
    hipEventRecord(start);

    // Pre-rotation
    int blockSize = 256;
    int numBlocks = (N4 + blockSize - 1) / blockSize;
    doPreRotationFused<<<numBlocks, blockSize>>>(dev_input, dev_input1, dev_f0, dev_f1, dev_t, N4, shift, stride, N2, sine);
    hipDeviceSynchronize();

    // ifft
    hipfftHandle plan;
    hipfftResult result = hipfftPlan1d(&plan, N4, HIPFFT_C2C, 2);
    if (result != HIPFFT_SUCCESS)
    {
        exit(EXIT_FAILURE);
    }

    // batch of 2
    result = hipfftExecC2C(plan,
                          (hipfftComplex *)dev_f0,
                          (hipfftComplex *)dev_fft_output,
                          HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    hipfftDestroy(plan);

    // ch 1
    var_t *c0_output_offset = dev_output + (overlap >> 1);
    var_t *c1_output_offset = dev_output1 + (overlap >> 1);
    CHECK_CUDA_ERROR(hipMemcpy(c0_output_offset, dev_fft_output, size_fft, hipMemcpyDeviceToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(c1_output_offset, (char *)dev_fft_output + size_fft, size_fft, hipMemcpyDeviceToDevice));
    CHECK_LAST_CUDA_ERROR(); // Check for errors after FFT execution
    hipDeviceSynchronize(); // Ensure all operations are complete

    // post-rotation and mirror
    int max_elements = max((N4 + 1) >> 1, overlap / 2);
    int numBlocksFused = (max_elements + blockSize - 1) / blockSize;
    postAndMirrorKernelFused<<<numBlocksFused, blockSize>>>(dev_output, dev_output1, dev_t, dev_window,
                                                           N2, N4, shift, sine, overlap);
    CHECK_LAST_CUDA_ERROR();
    hipDeviceSynchronize();
    
    // Stop GPU kernel timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, start, stop);

    // Update statistics
    total_kernel_time += kernel_time;
    call_count++;
    
    float avg_time = total_kernel_time / call_count;

    printf("MDCT Performance (#%d):\n", call_count);
    printf("  Current Kernel Time: %.3f ms\n", kernel_time);
    printf("  Average Kernel Time: %.3f ms\n", avg_time);
    
    // Copy final results and print
    CHECK_CUDA_ERROR(hipMemcpy(output[0], dev_output, size_output, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(output[1], dev_output1, size_output, hipMemcpyDeviceToHost));

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev_buf_ptr);
}

void cleanupCudaBuffers() {
    for (auto &it : dev_buf) {
        hipFree(it.second);
    }
    dev_buf.clear();
    for (auto &it : fft_buf) {
        cuda_fft_free(it.second);
    }
    fft_buf.clear();
}
