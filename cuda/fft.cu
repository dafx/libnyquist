#include <hipfft/hipfft.h>
#include <stdio.h>
#include "mdct_cuda.hpp"


// Initialization function
cuda_fft_state* cuda_fft_alloc(int nfft, int shift) {
    cuda_fft_state *state = (cuda_fft_state*)malloc(sizeof(cuda_fft_state));
    if (!state) {
        return NULL;
    }
    
    // Initialize all fields to 0
    memset(state, 0, sizeof(cuda_fft_state));
    
    state->nfft = nfft;
    state->shift = shift;
    
    // Create plan
    hipfftResult result = hipfftPlan1d(&state->plan, nfft, HIPFFT_C2C, 1);
    if (result != HIPFFT_SUCCESS) {
        free(state);
        return NULL;
    }
    
    state->initialized = 1;
    return state;
}

// Safe release function
void cuda_fft_free(cuda_fft_state *state) {
    if (state) {
        if (state->initialized) {
            // First synchronize all CUDA operations
            hipDeviceSynchronize();
            
            // Check and free device memory
            if (state->d_in) {
                hipFree(state->d_in);
                state->d_in = NULL;
            }
            
            if (state->d_out) {
                hipFree(state->d_out);
                state->d_out = NULL;
            }
            
            // Destroy plan
            if (state->plan) {
                hipfftDestroy(state->plan);
            }
            
            state->initialized = 0;
        }
        
        // Finally, free the structure
        free(state);
    }
}

// Function to execute FFT
int cuda_fft_execute(cuda_fft_state *state, 
                    const float *input,
                    float *output) {
    if (!state || !state->initialized) {
        return -1;
    }
    
    // Check input
    if (!input || !output) {
        return -2;
    }
    
    hipError_t error;
    
    // Copy input data
    error = hipMemcpy(state->d_in, input,
                      state->nfft * sizeof(hipfftComplex),
                      hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return -3;
    }
    
    // Execute FFT
    hipfftResult result = hipfftExecC2C(state->plan,
                                     (hipfftComplex *)state->d_in,
                                     (hipfftComplex *)state->d_out,
                                     HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) {
        return -4;
    }
    
    // Wait for GPU to complete
    hipDeviceSynchronize();
    
    // Copy results
    error = hipMemcpy(output, state->d_out,
                      state->nfft * sizeof(hipfftComplex),
                      hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        return -5;
    }
    
    return 0;
}